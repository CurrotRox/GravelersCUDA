#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>

#define N 5000000  // Number of roll instances. 
#define ROLLS 231  // Number of rolls per instance
#define THREADROLLS 200 // Each instance is running this many times. THREADROLLS * N should equal 1 billion

__global__ void rollAndFindMaxKernel(int* maxResult, int* d_numberOfIterations, unsigned long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < N) {
        //Init xorshift
        unsigned int state = seed ^ clock() + (idx + 1); //Ensure each thread starts with a new xorshift seed
        for (int j=0;j<THREADROLLS;j++){
            int count = 0;
            for (int i = 0; i < ROLLS; i++) {
                //Moved xorshift directly into the code. This sped it up a little bit
                state ^= state << 13;
                state ^= state >> 17;
                state ^= state << 5;
                //Now the actual counting looks like this
                if (state % 4 == 0) count++;
            }

            // Use atomic operation to update the maximum result, and iteration count
            atomicMax(maxResult, count);
            atomicAdd(d_numberOfIterations, 1);

        }
    }
}

int main() {

    //Time recording stuff
    float totalTime=0;
    float kernelTime=0;
    hipEvent_t start, stop, startKernel, stopKernel;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&startKernel);
    hipEventCreate(&stopKernel);
    hipEventRecord(start, 0);

    //Init both variables
    int* d_maxResult;
    int h_maxResult = 0;
    int* d_numberOfIterations;
    int h_numberOfIterations = 0;

    // Allocate memory on the device
    hipMalloc((void**)&d_maxResult, sizeof(int));
    hipMalloc(&d_numberOfIterations, sizeof(int));

    // Initialize the max result on the device
    hipMemcpy(d_maxResult, &h_maxResult, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_numberOfIterations, &h_numberOfIterations, sizeof(int), hipMemcpyHostToDevice);

    // Kernel setup
    int threadsPerBlock = 512;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;


    hipEventRecord(startKernel, 0);
    //Run the kernel (The actual simulation)
    rollAndFindMaxKernel << <blocksPerGrid, threadsPerBlock >> > (d_maxResult, d_numberOfIterations, time(NULL));

    hipEventRecord(stopKernel, 0);
    // Copy the max result back to the host
    hipMemcpy(&h_maxResult, d_maxResult, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&h_numberOfIterations, d_numberOfIterations, sizeof(int), hipMemcpyDeviceToHost);

    //Finishing time recording stuff
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&totalTime, start, stop);
    hipEventElapsedTime(&kernelTime, startKernel, stopKernel);

    // Print the results
    printf("Max ones: %d\n", h_maxResult);
    printf("Iterations: %d\n", h_numberOfIterations);
    printf("Total time (ms): %f\n", totalTime);
    printf("Total kernel time (ms): %f\n\n", kernelTime);

    // Free device memory
    hipFree(d_maxResult);
    hipFree(d_numberOfIterations);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipEventDestroy(startKernel);
    hipEventDestroy(stopKernel);

    
    return 0;
}
