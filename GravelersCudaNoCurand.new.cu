#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>

#define N 10000000  // Number of roll instances
#define ROLLS 231  // Number of rolls per instance


//Inserting xorshift32 onto device for hopefully faster RNG production
__device__ unsigned int xorshift32(unsigned int* state) {
    unsigned int x = *state;
    x ^= x << 13;
    x ^= x >> 17;
    x ^= x << 5;
    x ^= clock();
    *state = x;
    return x;
}

__global__ void rollAndFindMaxKernel(int* maxResult, int* d_numberOfIterations, unsigned long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < N) {
        unsigned int state = seed ^ clock() + (idx + 1);
        for (int j=0;j<100;j++){
            //Init xorshift

            int count = 0;
            for (int i = 0; i < ROLLS; i++) {
                int roll = xorshift32(&state) % 4;  // Generate a random number between 0 and 3
                if (roll == 0) count++;  // Increment if the roll is zero
            }

            // Use atomic operation to update the maximum result, and iteration count
            atomicMax(maxResult, count);
            atomicAdd(d_numberOfIterations, 1);
        }
    }
}

int main() {
    int* d_maxResult;
    int h_maxResult = 0;
    int* d_numberOfIterations;
    int h_numberOfIterations = 0;

    

    // Allocate memory on the device
    hipMalloc((void**)&d_maxResult, sizeof(int));
    hipMalloc(&d_numberOfIterations, sizeof(int));
    // Initialize the max result on the device
    hipMemcpy(d_maxResult, &h_maxResult, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_numberOfIterations, &h_numberOfIterations, sizeof(int), hipMemcpyHostToDevice);
    // Kernel setup
    int threadsPerBlock = 512;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    //Time recording stuff
    float totalTime=0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);


    rollAndFindMaxKernel << <blocksPerGrid, threadsPerBlock >> > (d_maxResult, d_numberOfIterations, time(NULL));
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&totalTime, start, stop);


    // Copy the max result back to the host
    hipMemcpy(&h_maxResult, d_maxResult, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&h_numberOfIterations, d_numberOfIterations, sizeof(int), hipMemcpyDeviceToHost);
    // Print the results
    printf("Max ones: %d\n", h_maxResult);
    printf("Iterations: %d\n", h_numberOfIterations);
    printf("Total time (ms): %f\n\n", totalTime);
    // Free device memory
    hipFree(d_maxResult);
    hipFree(d_numberOfIterations);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
